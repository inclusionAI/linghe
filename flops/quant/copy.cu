
#include <hip/hip_runtime.h>
#include <cstdio>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <stdio.h>
#include <iostream>

#define dtype __hip_bfloat16

__global__ void init_data_kernel(dtype *x, int N) {
    // int global_idx = blockIdx.x * 1024 + threadIdx.x;
    int global_idx = threadIdx.x;
    for (int i = global_idx; i < 1024; i += blockDim.x) {
        x[i] = __float2bfloat16(float(i));
        // x[i] = float(i);
        // if (threadIdx.x == 1 && blockIdx.x == 0) {
        // if (blockIdx.x == 0) {
        //       printf("bfx %f, x %f\n", __bfloat162float(x[i]), float(i));
        // }
    }
}

__global__ void cp_kernel(dtype *x, int N) {

  const int BYTES = 16;
  const int threads = 128;
  const int num_per_thread = BYTES / sizeof(dtype);
  __shared__ dtype smem[threads * num_per_thread];
  int index = threadIdx.x * num_per_thread;

  uint32_t smem_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(smem + index));
  dtype* glob_ptr = x + index;

  float A_frag[4];

  asm volatile(
    "{\n"
    // " cp.async.cg.shared.global [%0], [%1], %2, 8;\n" 
    " cp.async.cg.shared.global [%0], [%1], %2, 16;\n"
    // "cp.async.ca.shared.global.L2::128B [%0], [%1], 16;\n"
    "cp.async.commit_group;\n"
    "cp.async.wait_group 0\n;"
    "}\n" :: "r"(smem_ptr), "l"(glob_ptr), "n"(BYTES)
  );

  if (threadIdx.x == 1 && blockIdx.x == 0) {
    for (int i = 0; i < threads * num_per_thread; i++) {
        printf("smem %f \n", __bfloat162float(smem[i]));
    }
  }
  
  asm volatile (
    "ld.shared.v4.f32 {%0, %1, %2, %3}, [%4];\n"
    : "=f"(A_frag[0]), "=f"(A_frag[1]), "=f"(A_frag[2]), "=f"(A_frag[3])
    : "r"(smem_ptr)
  );

  __hip_bfloat162 a = __float2bfloat162_rn(A_frag[0]);
  __hip_bfloat162 b = __float2bfloat162_rn(A_frag[1]);
  __hip_bfloat162 c = __float2bfloat162_rn(A_frag[2]);
  __hip_bfloat162 d = __float2bfloat162_rn(A_frag[3]);

  if (threadIdx.x == 1 && blockIdx.x == 0) {
      __hip_bfloat16 low_bf16 = __low2bfloat16(a);
      __hip_bfloat16 high_bf16 = __high2bfloat16(a);
      // __nv_bfloat16 first_bf16 = a.x;
      printf("reg a.x %f \n", __bfloat162float(low_bf16));
      printf("reg a.y %f \n", __bfloat162float(high_bf16));
      // printf("reg a.x %f \n", __bfloat162float(a.x));
      // printf("reg a.y %f \n", __bfloat162float(a.y));
      printf("reg b.x %f \n", __bfloat162float(b.x));
      printf("reg b.y %f \n", __bfloat162float(b.y));
      printf("reg c.x %f \n", __bfloat162float(c.x));
      printf("reg c.y %f \n", __bfloat162float(c.y));
  }
  
}

int main() {
    const int N_DATA = 1024 * 1024;
    dtype *x;
    hipMalloc(&x, N_DATA * sizeof(dtype));
    int block = 256;
    int grid = 1024;
    init_data_kernel<<<grid, block>>>(x, N_DATA);

    cp_kernel<<<grid, 128>>>(x, N_DATA);  
    hipDeviceSynchronize();
    std::cout <<  hipGetErrorString( hipGetLastError() ) << std::endl;
    hipFree(x);
    return 0;
}